
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <algorithm>

#define NUM_THREADS 256


/*********************
 * GLOBAL VARIABLES *
*********************/
float k;
float starting_temp;
int dimension;
int timesteps;
int width, height, depth;
int num_fixed_points;
std::vector<float> fixed_points;


/*******************
 * HOST FUNCTIONS *
/******************/
void read_file(std::string file_name);                                                                                // Reads configuration file
float* alloc_host_mem(int width, int height, int depth, float temp);                                                  // Allocates space for a "simulated" array on the host  
float* alloc_device_mem(int width, int height, int depth);                                                            // Allocates space for a "simulated" array on the device
void print_grid(float* grid, int width, int height, int depth);                                                       // Prints the grid to a csv file
void set_fixed_temps_2D(std::vector<float> fixed_points, int num_fixed_points, float* grid, int width);               // Sets the fixed temp blocks in the grid (2D)
void set_fixed_temps_3D(std::vector<float> fixed_points, int num_fixed_points, float* grid, int width, int height);   // Sets the fixed temp blocks in the grid (3D)

/*********************
 * DEVICE FUNCTIONS *
/********************/
__global__ void update_grid_2D(float* d_old_grid, float* d_new_grid, int width, int height, float k);                 // Updates 2D grid (one timestep)
__global__ void update_grid_3D(float* d_old_grid, float* d_new_grid, int width, int height, int depth, float k);      // Updates 3D grid (one timestep)


/*********************
  ****** MAIN *******
/********************/
int main(int argc, char** argv) {
    // Read in the file to get the configurations
    read_file(argv[1]);

    // Allocate space for grids
    float* grid = alloc_host_mem(width, height, depth, starting_temp);      // HOST MEMORY (set blocks to default temp)
    float* d_new_grid = alloc_device_mem(width, height, depth);             // DEVICE MEMORY
    float* d_old_grid = alloc_device_mem(width, height, depth);             // DEVICE MEMORY
    // Size var for size of array
    int size = width*height*depth*sizeof(float);
    // Determine number of blocks to launch
    int NUM_BLOCKS = (int)((width*height*depth)/NUM_THREADS) + 1;

    // 2D timestep loop
    if (dimension == 2) {
        // Set the fixed temps (first time)
        set_fixed_temps_2D(fixed_points, num_fixed_points, grid, width);
        /*** Timestep Loop ***/
        for (int i = 0; i < timesteps; ++i) {
            // Copy host grid to device
            hipMemcpy(d_old_grid, grid, size, hipMemcpyHostToDevice);
            // Call device function (updates grid for this timestep)
            update_grid_2D<<<NUM_BLOCKS, NUM_THREADS>>>(d_old_grid, d_new_grid, width, height, k);
            // Copy device grid to host
            hipMemcpy(grid, d_new_grid, size, hipMemcpyDeviceToHost);
            // Set the fixed temps (reset every timestep)
            set_fixed_temps_2D(fixed_points, num_fixed_points, grid, width);
        }
    }
    // 3D timestep loop
    if (dimension == 3) {
        // Set the fixed temps (reset every timestep)
        set_fixed_temps_3D(fixed_points, num_fixed_points, grid, width, height);
        /*** Timestep Loop ***/
        for (int i = 0; i < timesteps; ++i) {
            // Copy host grid to device
            hipMemcpy(d_old_grid, grid, size, hipMemcpyHostToDevice);
            // Call device function (updates grid for this timestep)
            update_grid_3D<<<NUM_BLOCKS, NUM_THREADS>>>(d_old_grid, d_new_grid, width, height, depth, k);
            // Copy device grid to host
            hipMemcpy(grid, d_new_grid, size, hipMemcpyDeviceToHost);
            // Set the fixed temps (reset every timestep)
            set_fixed_temps_3D(fixed_points, num_fixed_points, grid, width, height);
        }
    }

    // Print out the results (to the output file)
    print_grid(grid, width, height, depth);

    // Free up that memory
    free(grid);
    hipFree(d_old_grid);
    hipFree(d_new_grid);

    return 0;
}


/*******************
 * HOST FUNCTIONS *
/******************/
// Read in the configuration file and populate global variables
void read_file(std::string file_name) {
    // Use std namespace for function calls (string, etc...)
    using namespace std;
    // Open a file
    ifstream fh(file_name);
    // find the first line ignore comments/empty lines
    string line;
    getline(fh, line);
    line.erase(remove(line.begin(), line.end(), ' '), line.end());
    while (line[0] == '#' || line.empty()) { getline(fh, line); line.erase(remove(line.begin(), line.end(), ' '), line.end()); }
    // get rid of possible whitespaces in the line
    line.erase(remove(line.begin(), line.end(), ' '), line.end());
    // Set the dimension (2 or 3)
    if (line.compare("2D") == 0)
        dimension = 2;
    else
        dimension = 3;

    //Loop through 5 more times to get the remaing arguments
    for (int i = 0; i < 5; ++i) {
        // Find the next line that is not a comment or empty
        getline(fh, line);
        // get rid of possible whitespaces in the line
        line.erase(remove(line.begin(), line.end(), ' '), line.end());
        while (line[0] == '#' || line.empty()) { getline(fh, line); line.erase(remove(line.begin(), line.end(), ' '), line.end());}
        // Use these for tokenizing line 3 and 5
        stringstream data(line);
        vector<string> tokens;
        string tok; 
        switch (i)
        {
            case 0:
                k = stof(line);
                break;
            case 1:
                timesteps = stoi(line);
                break;
            case 2:
                while(getline(data, tok, ',')) {
                    tokens.push_back(tok);
                }
                // Set width and height to the tokens
                width = stoi(tokens[0]);
                height = stoi(tokens[1]);
                if (dimension == 3) { depth = stoi(tokens[2]); }
                else { depth = 1; }
                break;
            case 3:
                starting_temp = stof(line);
                break;
            case 4:
                num_fixed_points++;
                while(getline(data, tok, ',')) {
                    fixed_points.push_back(stof(tok));
                }
                while(getline(fh, line)) {
                    if (line[0] != '#' && !line.empty()) {
                        stringstream tmp(line);
                        num_fixed_points++;
                        while (getline(tmp, tok, ',')) {
                            fixed_points.push_back(stof(tok));
                        }
                    }
                }
                break;
        }
    }

    fh.close();
}

// Allocates space for a "simulated" array on the host
float* alloc_host_mem(int width, int height, int depth, float temp) {
    float* grid = (float*)calloc(width*height*depth, sizeof(float));
    for (int i = 0; i < width*height*depth; ++i)
        grid[i] = temp;
    
    return grid;
}

// Allocates space for a "simulated" array on the device
float* alloc_device_mem(int width, int height, int depth) {
    float* d_grid; 
    hipMalloc((void**)&d_grid, width*height*depth*sizeof(float));

    return d_grid;
}

// Print the grid to the output file
void print_grid(float* grid, int width, int height, int depth) {
    // Open a file
    //int size = width*height;
    std::ofstream fh("heatOutput.csv");
    for (int d = 0; d < depth; ++d) {
        for (int i = 0; i < height; ++i) {
            for (int j = 0; j < width; ++j) {
                fh << grid[d*width*height + i*width + j];
                // Print a comma if youre not the last element
                if (j != width - 1) 
                    fh << ", ";
            }
            //Move to a new line each row
            fh << "\n";
        }
        // Put a blank line in between each slice
        fh << "\n\n";
    }
    fh.close();
}

// Take a grid and vector of fixed points and fill in the fixed temps (2D)
void set_fixed_temps_2D(std::vector<float> fixed_points, int num_fixed_points, float* grid, int width) {
    // Fill in fixed temp spots
    for (int i = 0; i < num_fixed_points; ++i) {
        // Extract the current fixed point
        int f_x = fixed_points[i*5 + 0];
        int f_y = fixed_points[i*5 + 1];
        int f_width = fixed_points[i*5 + 2];            
        int f_height = fixed_points[i*5 + 3];
        int f_temp = fixed_points[i*5 + 4];

        for (int j = 0; j < f_height; ++j) {
            for (int k = 0; k < f_width; ++k) {
                grid[(f_y+j)*width + f_x + k] = f_temp;
            }
        }
    }
}

// Take a grid and vector of fixed points and fill in the fixed temps (3D)
void set_fixed_temps_3D(std::vector<float> fixed_points, int num_fixed_points, float* grid, int width, int height) {
    // Fill in fixed temp spots
    for (int i = 0; i < num_fixed_points; ++i) {
        // Extract the current fixed point
        int f_x = fixed_points[i*7 + 0];
        int f_y = fixed_points[i*7 + 1];
        int f_z = fixed_points[i*7 + 2];
        int f_width = fixed_points[i*7 + 3];            
        int f_height = fixed_points[i*7 + 4];
        int f_depth = fixed_points[i*7 + 5];
        int f_temp = fixed_points[i*7 + 6];

        for (int d = 0; d < f_depth; ++d) {
            for (int j = 0; j < f_height; ++j) {
                for (int k = 0; k < f_width; ++k) {
                    grid[(f_z+d)*width*height + (f_y+j)*width + (f_x + k)] = f_temp;       // (f_z+d) + (f_y+j)*depth + (f_x + k)*height*depth
                }
            }
        }
    }
}

/*********************
 * DEVICE FUNCTIONS *
/********************/
// Temp diffusion function that updates the 2D grid each timestep
__global__ void update_grid_2D(float* d_old_grid, float* d_new_grid, int width, int height, float k) {
    // Get the current index in the device array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Make sure the index is within the array (blockDim is not always a perfect multiple)
    if (index < width * height) {
        // Extract the (x,y) cooridnates to check for edge cases
        int x, y;
        x = index % width;
        y = (int)(index/width);

        // Values for calculating next temp
        float T_old, T_top, T_bottom, T_left, T_right;
        T_old = d_old_grid[index];

        // Get T_left
        if (x != 0)
            T_left = d_old_grid[index - 1];
        else
            T_left = T_old;
        // Get T_top
        if (y != 0)
            T_top = d_old_grid[index - width] ;
        else
            T_top = T_old;
        // Get T_right
        if (x != width - 1)
            T_right = d_old_grid[index + 1];
        else
            T_right = T_old;
        // Get T_bottom
        if (y != height - 1)
            T_bottom = d_old_grid[index + width];
        else
            T_bottom = T_old;

        // Update d_new_grid with new value
        d_new_grid[index] = T_old + k*(T_top + T_bottom + T_left + T_right - 4*T_old);
    }
}

// Temp diffusion function that updates the 3D grid each timestep
__global__ void update_grid_3D(float* d_old_grid, float* d_new_grid, int width, int height, int depth, float k) {
    // Get the current index in the device array
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Make sure the index is within the array (blockDim is not always a perfect multiple)
    if (index < width*height*depth) {
        // Extract the (x,y,z) cooridnates to check for edge cases
        int x, y, z;
        z = (int)(index/(width*height));
        y = (int)((index-width*height*z)/width);
        x = (index-width*height*z) % width;

        // Values for calculating next temp
        float T_old, T_top, T_bottom, T_left, T_right, T_front, T_back;
        T_old = d_old_grid[index];

        // Get T_left
        if (x != 0)
            T_left = d_old_grid[index - 1];
        else
            T_left = T_old;
        // Get T_top
        if (y != 0)
            T_top = d_old_grid[index - width] ;
        else
            T_top = T_old;
        // Get T_right
        if (x != width - 1)
            T_right = d_old_grid[index + 1];
        else
            T_right = T_old;
        // Get T_bottom
        if (y != height - 1)
            T_bottom = d_old_grid[index + width];
        else
            T_bottom = T_old;
        // Get T_front
        if (z != 0)
            T_front = d_old_grid[index - width*height];
        else 
            T_front = T_old;
        // Get T_back
        if (z != depth - 1)
            T_back = d_old_grid[index + width*height];
        else
            T_back = T_old;

        // Update d_new_grid with new value
        d_new_grid[index] = T_old + k*(T_top + T_bottom + T_left + T_right + T_front + T_back - 6*T_old);
    }
}